#include "hip/hip_runtime.h"
#include <iostream>
#include "../CollisionShape/CollisionShape.h"
#include "./Detection.cuh"

typedef struct Vec2 {
    double x, y;
} Vec2;

__device__ bool cmpVec(Vec2* first, Vec2* other)
{
    const float TOLERANCE = 0.0001;
    auto a = std::abs(first->x - other->x) < TOLERANCE;
    auto b = std::abs(first->y - other->y) < TOLERANCE;
    return a && b;
}

struct CollisionDetail {
    bool collision = false;
    double leastOverlap = 1000000000;
    double greatestOverlap = 0.f;
    Vec2 leastAxis{};
};

typedef struct CPoint {
    float dist{};
    Vec2 point{};
    bool contains{};
} CPoint;

__device__ void getLine(Vec2 p1, Vec2 p2, double &a, double &b, double &c)
{
    // (x- p1X) / (p2X - p1X) = (y - p1Y) / (p2Y - p1Y)
    a = p1.y - p2.y;
    b = p2.x - p1.x;
    c = p1.x * p2.y - p2.x * p1.y;
}

__device__ CPoint dist(Vec2 p, Vec2 start, Vec2 end)
{
    Vec2 edge{end.x - start.x, end.y - start.y};
    Vec2 pointDir{p.x - start.x, p.y - start.y};
    auto proj = pointDir.x * edge.x + pointDir.y * edge.y;
    auto contains = proj >= 0 && proj <= (edge.x * edge.x + edge.y * edge.y);

    double a, b, c;
    // ax + by + c = 0

    getLine(start, end, a, b, c);

    auto distance = (float)(abs(a * p.x + b * p.y + c) / sqrt(a * a + b * b));
    // Calculate the coordinates of the closest point on the line
    auto closestPointX = (float)((b * (b * p.x - a * p.y) - a * c) / (a * a + b * b));
    auto closestPointY = (float)((a * (-b * p.x + a * p.y) - b * c) / (a * a + b * b));

    return {distance, {closestPointX, closestPointY}, contains};
}

__device__ void findMinDistance(
        Vec2** globals,
        const unsigned int sizes[2],
        bool inverse,
        Vec2* contact1,
        Vec2* contact2,
        float* minDistSq,
        int* contactCount
)
{
    auto first = 0;
    auto second = 1;
    if (inverse)
    {
        first = 1;
        second = 0;
    }
    auto globalA = globals[first];
    auto globalB = globals[second];

    auto verticesA = sizes[first];
    auto verticesB = sizes[second];

    for (int i = 0; i < verticesA; i++) {
        auto vertexA = globalA[i];

        for (int j = 0; j < verticesB; j++) {
            auto bStart = globalB[j];
            auto bEnd = globalB[(j + 1) % verticesB];

            auto cp = dist(vertexA, bStart, bEnd);

            if (!cp.contains)
                continue;

            if (abs(cp.dist - *minDistSq) < 0.0001)
            {
                if (cmpVec(&cp.point, contact1)) continue;

                *contact2 = cp.point;
                *contactCount = 2;
            }
            else if (cp.dist < *minDistSq)
            {
                *minDistSq = cp.dist;
                *contactCount = 1;
                *contact1 = cp.point;
            }
        }
    }
}

__device__ CollidingPoints findContactPoints(
        Vec2** globals,
        unsigned int sizes[2]
)
{
    auto contact1 = Vec2();
    auto contact2 = Vec2();
    auto contactCount = 0;

    float minDistSq = INT8_MAX;

    findMinDistance(
            globals,
            sizes,
            false,
            &contact1,
            &contact2,
            &minDistSq,
            &contactCount
    );

    findMinDistance(
            globals,
            sizes,
            true,
            &contact1,
            &contact2,
            &minDistSq,
            &contactCount
    );

    return CollidingPoints{*(Vector2*)(&contact1), *(Vector2*)(&contact2), contactCount};
}

__device__ void getProjection(Vec2 axis, Vec2* vertices, unsigned int length, Vec2* result)
{
    auto vertex = vertices[0];
    double min = axis.x * vertex.x + axis.y * vertex.y;
    double max = min;
    for (int i = 1; i < length; i++) {
        vertex = vertices[i];
        double p = axis.x * vertex.x + axis.y * vertex.y;
        if (p < min) {
            min = p;
        } else if (p > max) {
            max = p;
        }
    }
    result->x = min;
    result->y = max;
}

__global__ void getCollisionAxis(
        Vec2* axes,
        Vec2* vertices1,
        unsigned int size1,
        Vec2* vertices2,
        unsigned int size2,
        CollisionDetail* details
        )
{
    auto axis = axes[threadIdx.x];
    auto detail = &details[threadIdx.x];

    Vec2 p1;
    getProjection(axis, vertices1, size1, &p1);
    Vec2 p2;
    getProjection(axis, vertices2, size2, &p2);

    bool overlap = p1.x < p2.y && p2.x < p1.y;
    if (!overlap) {
        detail->collision = false;
    }
    else {
        detail->collision = true;
        double minOverlap = max(p1.x, p2.x);
        double maxOverlap = min(p1.y, p2.y);
        Vec2 newOverlap = {minOverlap, maxOverlap};
        auto nO = abs(newOverlap.x - newOverlap.y);

        detail->leastOverlap = nO;
        auto a = axis;
        if (p1.y > p2.y)
            a.y *= -1;
        if (p1.x > p2.x)
            a.x *= -1;

        detail->greatestOverlap = nO;
        detail->leastAxis = a;
    }
}

typedef struct CudaShape {
    unsigned int pointCount;
    Vec2* points;
    float* transform;
    ShapeType type;
} CudaShape;

__global__ void getGlobalVertices(CudaShape* shapes, const unsigned int sizes[2], Vec2 offset, Vec2** vertices)
{
    auto i = blockIdx.x;
    auto s = shapes[i];
    if (threadIdx.x >= sizes[i])
        return;
    auto vertex = s.points[threadIdx.x];
    auto matrix = s.transform;

    Vec2 res;
    res.x = (matrix[0] * vertex.x) + (matrix[4] * vertex.y) + matrix[12];
    res.y = (matrix[1] * vertex.x) + (matrix[5] * vertex.y) + matrix[13];

    if (i == 0)
    {
        res.x += offset.x;
        res.y += offset.y;
    }

    vertices[i][threadIdx.x].x = res.x;
    vertices[i][threadIdx.x].y = res.y;
}

__global__ void getAxes(Vec2** vertices, const unsigned int sizes[2], Vec2** axes)
{
    const unsigned int i = blockIdx.x;
    auto v = vertices[i];
    unsigned int length = sizes[i];
    const unsigned int j = threadIdx.x;

    //Getting vertex
    auto v1 = v[j];
    //Getting other vertex
    auto v2 = v[j+1 == length ? 0 : j+1];

    Vec2 edge;
    edge.x = v1.x - v2.x;
    edge.y = v1.y - v2.y;

    Vec2 normal;
    normal.x = -edge.y;
    normal.y = edge.x;

    double l = sqrt(normal.x * normal.x + normal.y * normal.y);
    normal.x /= l;
    normal.y /= l;

    //Normalizing to get accurate projections
    axes[i][j] = normal;
}

__global__ void getFinalDetail(Colliding res[2], CollisionDetail* details, unsigned int length, unsigned int index)
{
    CollisionDetail final;
    final.collision = true;

    for (int i = 0; i < length; i++) {
        auto detail = details[i];

        if (!detail.collision)
        {
            final.collision = false;
            break;
        }

        if (detail.leastOverlap < final.leastOverlap)
        {
            final.leastOverlap = detail.leastOverlap;
            final.leastAxis = detail.leastAxis;
        }
        if (detail.greatestOverlap > final.greatestOverlap)
            final.greatestOverlap = detail.greatestOverlap;
    }

    res[index].collision = final.collision;
    res[index].penetration = final.greatestOverlap;
    res[index].overlap = final.leastOverlap;
    res[index].normal.x = final.leastAxis.x;
    res[index].normal.y = final.leastAxis.y;
}

__global__ void getCollision(Vec2** axes, Vec2** vertices, unsigned int sizes[2], Colliding res[2])
{
    const unsigned int index = blockIdx.x;
    auto a = axes[index];
    auto length = sizes[index];
    auto details = (CollisionDetail*)malloc(sizeof(CollisionDetail) * length);
    getCollisionAxis<<<1, length>>>(a, vertices[0], sizes[0], vertices[1], sizes[1], details);

    getFinalDetail<<<1,1>>>(res, details, length, index);
}

__global__ void done(Vec2** globals, unsigned int sizes[2], Colliding* cs, unsigned int i, Colliding* results)
{
    auto c1 = cs[0];
    auto c2 = cs[1];
    auto colliding = c1.collision && c2.collision;

    auto obj = (Colliding*)malloc(sizeof(Colliding));
    obj->collision = colliding;

    obj->overlap = c1.overlap;
    obj->normal = c1.normal;

    obj->penetration = c1.penetration;

    obj->collidingPoints = findContactPoints(globals, sizes);

    results[i] = *obj;

    free(obj);
}

__global__ void freeAll(Vec2** axes, Vec2** globals, unsigned int sizes[2], CudaShape* shapes, Colliding* cs)
{
    free(axes);
    free(globals);
    free(sizes);
    free(shapes);
    free(cs);
}

__global__ void checkContinuous(int m, Vector2 direction, CudaShape first, CudaShape second, Colliding* results)
{
    auto i = threadIdx.x;
    float j = (float)i / (float)m;

    //auto pos = startPos + j * direction;
    Vec2 pos{0,0};

    pos.x += j * direction.x;
    pos.y += j * direction.y;

    /*for (int k = 0; k < first.pointCount; ++k) {
        first.points[k].x += (float)pos.x;
        first.points[k].y += (float)pos.y;
    }*/

    //satCollision(ShapeType::RECTANGLE, &first, &second, colliding);

    if (first.type == CIRCLE && second.type == CIRCLE)
    {
        printf("Circle\n");
        return;
    }
    else if (first.type == RECTANGLE && second.type == RECTANGLE)
    {
        auto maxSize = max(first.pointCount, second.pointCount);
        auto sizes = new unsigned int[2];
        sizes[0] = first.pointCount;
        sizes[1] = second.pointCount;

        Vec2** globals = (Vec2**)malloc(sizeof(Vec2*) * 2);
        globals[0] = (Vec2*)malloc(sizeof(Vec2) * first.pointCount);
        globals[1] = (Vec2*)malloc(sizeof(Vec2) * second.pointCount);

        auto shapes = new CudaShape[2];
        shapes[0] = first;
        shapes[1] = second;
        getGlobalVertices<<<2, maxSize>>>(shapes, sizes, pos, globals);

        //Get axes
        Vec2** axes = (Vec2**)malloc(sizeof(Vec2*) * 2);
        axes[0] = (Vec2*)malloc(sizeof(Vec2)*first.pointCount);
        axes[1] = (Vec2*)malloc(sizeof(Vec2)*second.pointCount);
        getAxes<<<2, maxSize>>>(globals, sizes, axes);


        auto cs = (Colliding*)malloc(sizeof(Colliding) * 2);
        getCollision<<<2,1>>>(axes, globals, sizes, cs);

        done<<<1,1>>>(globals, sizes, cs, threadIdx.x, results);

        freeAll<<<1, 1>>>(axes, globals, sizes, shapes, cs);
    }
}

Colliding checkContinuousWrapper(Vector2 startPos, Vector2 direction, sf::Shape* s1, sf::Shape* s2) {
    int device = 0;
    hipDeviceProp_t deviceProp{};
    hipGetDeviceProperties(&deviceProp, hipGetDevice(&device));

    if(!(deviceProp.major > 3 || (deviceProp.major == 3 && deviceProp.minor >=5)))
    {
        printf("GPU %d - %s does not support CUDA Dynamic Parallelism\n Exiting.", device, deviceProp.name);
        exit(1);
    }

    CudaShape first;
    auto s = s1->getPointCount();
    first.pointCount = s;
    first.type = RECTANGLE;

    hipMalloc(&first.points, sizeof(Vector2) * s);
    for (int i = 0; i < s; i++) {
        auto p = (Vector2)s1->getPoint(i);
        hipMemcpy(&first.points[i], &p, sizeof(Vector2), hipMemcpyHostToDevice);
    }

    const float* matrix = s1->getTransform().getMatrix();
    hipMalloc(&first.transform, sizeof(float) * 16);
    for (int i = 0; i < 16; i++) {
        hipMemcpy(&first.transform[i], &matrix[i], sizeof(float), hipMemcpyHostToDevice);
    }

    CudaShape second;
    s = s2->getPointCount();
    second.pointCount = s;
    second.type = RECTANGLE;

    hipMalloc(&second.points, sizeof(Vector2) * s);
    for (int i = 0; i < s; i++) {
        auto p = (Vector2)s2->getPoint(i);
        hipMemcpy(&second.points[i], &p, sizeof(Vector2), hipMemcpyHostToDevice);
    }

    hipMalloc(&second.transform, sizeof(float) * 16);
    matrix = s2->getTransform().getMatrix();
    for (int i = 0; i < 16; i++) {
        hipMemcpy(&second.transform[i], &matrix[i], sizeof(float), hipMemcpyHostToDevice);
    }

    const float precision = .25f;
    const int max = (int)(1 / precision);

    Colliding* results;
    hipMalloc(&results, sizeof(Colliding) * max);
    checkContinuous<<<1, max>>>(max, direction, first, second, results);

    auto* r = (Colliding*)malloc(sizeof(Colliding) * max);
    hipMemcpy(r, results, sizeof (Colliding) * max, hipMemcpyDeviceToHost);

    hipFree(&first);
    hipFree(&second);
    hipFree(&results);

    Vector2 old = startPos;
    for (int i = 0; i < max; ++i) {
        auto c = r[i];
        if (c.collision) {
            s1->setPosition((sf::Vector2f)old);
            return c;
        }
        old = (startPos + direction * ((float)(i-1) / (float)max));
    }

    return r[0];
}